
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <sys/time.h>

#define BLOCK_SIZE 256
#define N 1000000

double get_clock() {
    struct timeval tv; 
    int ok;
    ok = gettimeofday(&tv, (void *) 0);
    if (ok<0) { printf("gettimeofday error"); }
    return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

__global__ void scan(int *in, int *out){
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < N) {
        int result = 0;
        for (int i = 0; i <= index; i++) {
            result += in[i]; 
        }
        out[index] = result;
    }
}

int main(void)
{
    // memory allocation
    int *in, *out;
    hipMallocManaged(&in, N * sizeof(int));
    hipMallocManaged(&out, N * sizeof(int));

    // array initialization
    for (int i = 0; i < N; i++) {
        in[i] = 1;
    }

    // start timer
    double t0 = get_clock();

    // Run kernel
    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    scan<<<numBlocks, BLOCK_SIZE>>>(in, out);

    // printf("%s\n", cudaGetErrorString(cudaGetLastError()));

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // stop timer
    double t1 = get_clock();

    // result
    printf("%d\n", out[N - 1]);
    printf("time per call: %f ns\n", (1000000000.0 * (t1 - t0) / N));

    // Free memory
    hipFree(in);
    hipFree(out);

    return 0;
}
